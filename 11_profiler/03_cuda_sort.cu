#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

__global__ void fillBucket(int* key, int *bucket, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=n) return;
  atomicAdd(&bucket[key[i]], 1);
}

__global__ void scanBucket(int *bucket, int *offset, int *buffer, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=range) return;
  grid_group grid = this_grid();
  offset[i] = bucket[i];
  for(int j=1; j<range; j<<=1) {
    buffer[i] = offset[i];
    grid.sync();
    if(i>=j) offset[i] += buffer[i-j];
    grid.sync();
  }
  offset[i] -= bucket[i];
}

__global__ void fillKey(int *key, int *bucket, int *offset, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=range) return;
  int j = offset[i];
  for (; bucket[i]>0; bucket[i]--)
    key[j++] = i;
}

int main() {
  int n = 10000000;
  int m = 256;
  int range = 100000;
  int *key, *bucket, *offset, *buffer;
  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));
  hipMallocManaged(&offset, range*sizeof(int));
  hipMallocManaged(&buffer, range*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
  }
  for (int i=0; i<range; i++)
    bucket[i] = 0;
  fillBucket<<<(n+m-1)/m,m>>>(key, bucket, n);
  void *args[] = {(void *)&bucket,  (void *)&offset, (void *)&buffer, (void*)&range};
  hipLaunchCooperativeKernel((void*)scanBucket, (range+m-1)/m, m, args);
  fillKey<<<(range+m-1)/m,m>>>(key, bucket, offset, range);
  hipDeviceSynchronize();
  for (int i=1; i<n; i++)
    assert(key[i] >= key[i-1]);
  hipFree(key);
  hipFree(bucket);
  hipFree(offset);
  hipFree(buffer);
}
