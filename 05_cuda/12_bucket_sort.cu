
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void bucketsort(int* key,int* bucket,int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>N) return;
  atomicAdd(&bucket[key[i]],1);
  __syncthreads();

  for(int accumulate=0,val=0;accumulate<=i;val++){
      key[i]=val;
      accumulate+=bucket[val];
  }
}

int main() {
  int N = 100;
  const int M=64;
  int range = 5;

  int *key;
  int *bucket;
  hipMallocManaged(&key,N*sizeof(int));
  for (int i=0; i<N; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  hipMallocManaged(&bucket,range*sizeof(int));
  for(int i=0;i<range;i++)bucket[i]=0;

  bucketsort<<<(N+M-1)/M,M>>>(key,bucket,N);

  hipDeviceSynchronize();


  for (int i=0; i<N; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}
